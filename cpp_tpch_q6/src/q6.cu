#include <iostream>
#include <chrono>
#include <vector>
#include <string>

#include "data_types.hpp"
#include "parser.hpp"
#include "gpu_kernels.h"

#define LINEITEM_PATH "data/lineitem.tbl"

void check_cpu(int n, double* l_quantity, int* l_shipdate, double* l_discount)
{
  for (int i = 0; i < n; i++) {
    bool valid_date = (l_shipdate[i] >= 727841 && l_shipdate[i] <= 727841);
    bool valid_quantity = (l_quantity[i] < 24.0);
    bool valid_discount = (l_discount[i] >= 0.05 && l_discount[i] < 0.07);
    l_quantity[i] = (valid_date && valid_quantity && valid_discount) ? 1 : 0;
  }
}

void multiply_cpu(int n, double* l_quantity, double* l_extendedprice, double* l_discount)
{
  for (int i = 0; i < n; i++) {
    l_extendedprice[i] = (l_quantity[i]) ? l_extendedprice[i]*l_discount[i] : 0;
  }
}

int main(int argc, char** argv)
{
  float r = 1.0;
  bool overwrite_file = false;
  if (argc > 1) {
    r = atof(argv[1]);
    std::cout << "Ratio: " << r << std::endl;
    if (argc > 2)
    {
      std::string str(argv[2]);
      if (str == "overwrite") overwrite_file = true;
    }
  } else { std::cout << "Ratio set to default: " << r << std::endl; }
  
  LineItem lineitem;
  Parser p;
  p.parse(LINEITEM_PATH, lineitem, overwrite_file);

  std::cout << "Starting program" << std::endl;
  double* l_quantity;
  double* l_extendedprice;
  double* l_discount;
  int* l_shipdate;
  int N = *lineitem.size;

  // Allocate Unified Memory – accessible from CPU or GPU
  std::cout << "Allocating Memory" << std::endl;
  hipMallocManaged(&l_quantity, N*sizeof(double));
  hipMallocManaged(&l_extendedprice, N*sizeof(double));
  hipMallocManaged(&l_discount, N*sizeof(double));
  hipMallocManaged(&l_shipdate, N*sizeof(int));
  
  std::cout << "Initializing values" << std::endl;
  for (int i = 0; i < N; i++) {
    l_quantity[i] = lineitem.l_quantity[i];
    l_extendedprice[i] = lineitem.l_extendedprice[i];
    l_discount[i] = lineitem.l_discount[i];
    l_shipdate[i] = lineitem.l_shipdate[i];
  }

  int N_cpu = N*r;
  int N_gpu = N*(1-r);
  int blockSize = 256;
  int numBlocks = (N_gpu + blockSize - 1) / blockSize;
  std::cout << "cpu:gpu ratio: " << r << ":" << (1-r) << std::endl;

  auto start1 = std::chrono::steady_clock::now(); 

  std::cout << "Running kernels" << std::endl;
  check_cpu(N_cpu, l_quantity, l_shipdate, l_discount);
  check<<<numBlocks, blockSize>>>(N_gpu, l_quantity+N_cpu, l_shipdate+N_cpu, l_discount+N_cpu);
  hipDeviceSynchronize();

  auto total1 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start1).count();
  auto start2 = std::chrono::steady_clock::now(); 

  multiply_cpu(N_cpu, l_quantity, l_extendedprice, l_discount);
  multiply<<<numBlocks, blockSize>>>(N_gpu, l_quantity+N_cpu, l_extendedprice+N_cpu, l_discount+N_cpu);
  hipDeviceSynchronize();

  auto total2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start2).count();
  auto total3 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start1).count();
  std::cout << "Total time check: " << total1 << " ms" << std::endl;
  std::cout << "Total time multiply: " << total2 << " ms" << std::endl;
  std::cout << "Total time : " << total3 << " ms" << std::endl;

  // Read out 'query result'
  int amount = 0;
  double result = 0;
  for (int i = 0; i < N; i++)
  {
    if (l_extendedprice[i]) 
    {
      amount++;
      result += l_extendedprice[i];
    }
  }
  std::cout << "Query hit amount: " << amount << std::endl;
  std::cout << "Total tuples: " << N << std::endl;
  std::cout << std::fixed <<  "Result: " << result << std::endl;

  // Free memory
  hipFree(l_discount);
  hipFree(l_extendedprice);
  hipFree(l_shipdate);
  hipFree(l_discount);
  
  return 0;
}
