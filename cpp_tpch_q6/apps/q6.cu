#include <iostream>
#include <math.h>
#include <random>
#include "gpu_kernels.h"

// CPU function
void check_cpu(int n, int* a, int* b, int* c, int* d) {
  for (int i = 0; i < n; i++) {
    bool condition = (a[i]>50 && b[i]>50 && c[i]>50 && d[i]>50); // Mock condition
    a[i] = condition ? 1 : 0;
  }
}

int main(int argc, char** argv)
{
  float r = 0.f;
  try {
    r = atof(argv[1]);
    std::cout << "Ratio: " << r << std::endl;
  } catch (...) { std::cout << "Ratio set to default: " << r << std::endl; }

  std::cout << "Starting program" << std::endl;
  int N = 1<<20;
  int* l_shipdate;
  int* l_quantity;
  int* l_extendedprice;
  int* l_discount;

  // Allocate Unified Memory – accessible from CPU or GPU
  std::cout << "Allocating Memory" << std::endl;
  hipMallocManaged(&l_extendedprice, N*sizeof(int));
  hipMallocManaged(&l_discount, N*sizeof(int));
  hipMallocManaged(&l_quantity, N*sizeof(int));
  hipMallocManaged(&l_shipdate, N*sizeof(int));

  // initialize rng
  std::random_device dev;
  std::mt19937 rng(dev());
  std::uniform_int_distribution<std::mt19937::result_type> generateRandomInt(1, 100);

  // initialize x and y arrays on the host
  std::cout << "Initializing values" << std::endl;
  for (int i = 0; i < N; i++) {
    l_extendedprice[i] = generateRandomInt(rng);
    l_discount[i] = generateRandomInt(rng);
    l_shipdate[i] = generateRandomInt(rng);
    l_quantity[i] = generateRandomInt(rng);
  }

  int N_cpu = N*r;
  int N_gpu = N*(1-r);
  std::cout << "cpu:gpu ratio: " << N_cpu << ":" << N_gpu << std::endl;

  check_cpu(N_cpu, l_quantity, l_shipdate, l_extendedprice, l_discount);

  // Run kernel on the GPU
  std::cout << "Running kernels" << std::endl;
  int blockSize = 256;
  int numBlocks = (N_gpu + blockSize - 1) / blockSize;
  check<<<numBlocks, blockSize>>>(N_gpu, l_quantity, l_shipdate, l_extendedprice, l_discount);
  hipDeviceSynchronize();  // Is this necessary?

  multiply<<<numBlocks, blockSize>>>(N, l_quantity, l_extendedprice, l_discount);
  hipDeviceSynchronize();

  // Read out 'query result'
  int amount = 0;
  for (int i = 0; i < N; i++) if (l_extendedprice[i]) amount++;
  std::cout << "Amount: " << amount << std::endl;
  std::cout << "N: " << N << std::endl;

  // Free memory
  hipFree(l_discount);
  hipFree(l_extendedprice);
  hipFree(l_shipdate);
  hipFree(l_discount);
   
  return 0;
}
