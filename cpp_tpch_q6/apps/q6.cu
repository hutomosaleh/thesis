#include <iostream>
#include <iterator>
#include <istream>
#include <fstream>
#include <chrono>
#include <math.h>
#include <random>
#include <sstream>
#include <string>
#include "gpu_kernels.h"

#define QUANTITY 4
#define EXTENDED_PRICE 5
#define DISCOUNT 6
#define SHIPDATE 10
#define NUM_COLUMN 16
#define LINEITEM_PATH "cpp_tpch_q6/data/lineitem.tbl"
#define DELIMITER '|'

void check_cpu(int n, double* l_quantity, int* l_shipdate, double* l_discount)
{
  for (int i = 0; i < n; i++) {
    bool valid_date = (l_shipdate[i] >= 726350 && l_shipdate[i] <= 729313);
    bool valid_quantity = (l_quantity[i] < 70.0);
    bool valid_discount = (l_discount[i] >= 0.01 && l_discount[i] < 0.08);
    l_quantity[i] = (valid_date && valid_quantity && valid_discount) ? 1 : 0;
  }
}

void multiply_cpu(int n, double* l_quantity, double* l_extendedprice, double* l_discount)
{
  for (int i = 0; i < n; i++) {
    l_extendedprice[i] = (l_quantity[i]) ? l_extendedprice[i]*l_discount[i] : 0;
  }
}

template<class T>
void vec2ptr(std::vector<T> vec, T* ptr, int size)
{
  for (int i=0; i<size; i++) {
    ptr[i] = vec[i];
  }
}

template<typename T>
void bin2ptr(const char* filename, T **ptr)
{
    // open the file:
    std::streampos fileSize;
    std::ifstream file(filename, std::ios::binary);

    // get its size:
    file.seekg(0, std::ios::end);
    fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    // read the data
    char *temp = new char[fileSize];
    file.read(temp, fileSize);
    *ptr = reinterpret_cast<T*>(temp);
}

struct LineItem {
  double* l_quantity;
  double* l_extendedprice;
  double* l_discount;
  int* l_shipdate;
  int* size;
};

int dtoi(std::string str) {
  std::istringstream date(str);
  std::string time;
  int result = 0;
  int count = 0;
  while (getline(date, time, '-')) {
    int multiplier = 1;
    if (count==0) multiplier=365;
    if (count==1) multiplier=30;
    ++count;
    result += stoi(time)*multiplier;
  }
  return result;
}

void parse_lineitem(std::string path, LineItem& record, bool overwrite_file)
{ 
  bool file_exists = true;
  if (overwrite_file)
  {
    std::ofstream l_quantity ("cpp_tpch_q6/data/l_quantity.bin", std::ios::binary | std::ios::app);
    std::ofstream l_extendedprice ("cpp_tpch_q6/data/l_extendedprice.bin", std::ios::binary | std::ios::app);
    std::ofstream l_discount ("cpp_tpch_q6/data/l_discount.bin", std::ios::binary | std::ios::app);
    std::ofstream l_shipdate ("cpp_tpch_q6/data/l_shipdate.bin", std::ios::binary | std::ios::app);
    std::ofstream size ("cpp_tpch_q6/data/size.bin", std::ios::binary | std::ios::app);
    std::cout << "Parsing lineitem" << std::endl;

    std::fstream buffer(path);
    std::string line;
    int record_size = 0;
    while (getline(buffer, line)) {
      std::istringstream row(line);
      std::string field;
      int column;
      while (getline(row, field, DELIMITER)) {
        if (column==QUANTITY) {
          double q = std::stod(field);
          l_quantity.write(reinterpret_cast<char*>(&q), sizeof(double));
        } else if (column==EXTENDED_PRICE) {
          double q = std::stod(field);
          l_extendedprice.write(reinterpret_cast<char*>(&q), sizeof(double));
        } else if (column==DISCOUNT) {
          double q = std::stod(field);
          l_discount.write(reinterpret_cast<char*>(&q), sizeof(double));
        } else if (column==SHIPDATE) {
          int q = dtoi(field);
          l_shipdate.write(reinterpret_cast<char*>(&q), sizeof(int));
        }
        ++column;
        if (column==NUM_COLUMN) {
          column = 0;
          continue;
        }
      }
      ++record_size;
    }
    size.write(reinterpret_cast<char*>(&record_size), sizeof(int));
  }
  else
  {
    std::cout << "Table already parsed in binary, using that instead." << std::endl;
  }

  // Write binary to variables
  bin2ptr("cpp_tpch_q6/data/l_quantity.bin", &record.l_quantity);
  bin2ptr("cpp_tpch_q6/data/l_extendedprice.bin", &record.l_extendedprice);
  bin2ptr("cpp_tpch_q6/data/l_discount.bin", &record.l_discount);
  bin2ptr("cpp_tpch_q6/data/l_shipdate.bin", &record.l_shipdate);
  bin2ptr("cpp_tpch_q6/data/size.bin", &record.size);
}

int main(int argc, char** argv)
{
  float r = 1.0;
  bool overwrite_file = false;
  if (argc > 1) {
    r = atof(argv[1]);
    std::cout << "Ratio: " << r << std::endl;
    if (argc > 2)
    {
      std::string str(argv[2]);
      if (str == "overwrite") overwrite_file = true;
    }
  } else { std::cout << "Ratio set to default: " << r << std::endl; }
  
  LineItem lineitem;
  parse_lineitem(LINEITEM_PATH, lineitem, overwrite_file);

  std::cout << "Starting program" << std::endl;
  double* l_quantity;
  double* l_extendedprice;
  double* l_discount;
  int* l_shipdate;
  int N = *lineitem.size;

  std::cout << "Size: " << N << std::endl;
  std::cout << "l_quantity: " << lineitem.l_quantity[0] << std::endl;

  // Allocate Unified Memory – accessible from CPU or GPU
  std::cout << "Allocating Memory" << std::endl;
  hipMallocManaged(&l_quantity, N*sizeof(double));
  hipMallocManaged(&l_extendedprice, N*sizeof(double));
  hipMallocManaged(&l_discount, N*sizeof(double));
  hipMallocManaged(&l_shipdate, N*sizeof(int));
  
  std::cout << "Initializing values" << std::endl;
  for (int i = 0; i < N; i++) {
    l_quantity[i] = lineitem.l_quantity[i];
    l_extendedprice[i] = lineitem.l_extendedprice[i];
    l_discount[i] = lineitem.l_discount[i];
    l_shipdate[i] = lineitem.l_shipdate[i];
  }

  int N_cpu = N*r;
  int N_gpu = N*(1-r);
  int blockSize = 256;
  int numBlocks = (N_gpu + blockSize - 1) / blockSize;
  std::cout << "cpu:gpu ratio: " << r << ":" << (1-r) << std::endl;

  auto start = std::chrono::steady_clock::now(); 

  std::cout << "Running kernels" << std::endl;
  check_cpu(N_cpu, l_quantity, l_shipdate, l_discount);
  check<<<numBlocks, blockSize>>>(N_gpu, l_quantity+N_cpu, l_shipdate+N_cpu, l_discount+N_cpu);
  hipDeviceSynchronize();

  multiply_cpu(N_cpu, l_quantity, l_extendedprice, l_discount);
  multiply<<<numBlocks, blockSize>>>(N_gpu, l_quantity+N_cpu, l_extendedprice+N_cpu, l_discount+N_cpu);
  hipDeviceSynchronize();

  auto total = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start).count();
  std::cout << "Total time: " << total << " ms" << std::endl;

  // Read out 'query result'
  int amount = 0;
  for (int i = 0; i < N; i++) if (l_extendedprice[i]) amount++;
  std::cout << "Amount: " << amount << std::endl;
  std::cout << "N: " << N << std::endl;

  // Free memory
  hipFree(l_discount);
  hipFree(l_extendedprice);
  hipFree(l_shipdate);
  hipFree(l_discount);
  
  return 0;
}
