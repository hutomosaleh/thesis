#include <iostream>
#include <fstream>
#include <chrono>
#include <math.h>
#include <random>
#include <sstream>
#include <string>
#include "gpu_kernels.h"

#define QUANTITY 5
#define EXTENDED_PRICE 6
#define DISCOUNT 7
#define SHIPDATE 11
#define NUM_COLUMN 16
#define LINEITEM_PATH "cpp_tpch_q6/data/lineitem.tbl"
#define DELIMITER '|'

// CPU function
void check_cpu(int n, int* a, int* b, int* c, int* d)
{
  for (int i = 0; i < n; i++) {
    bool condition = (a[i]>50 && b[i]>50 && c[i]>50 && d[i]>50); // Mock condition
    a[i] = condition ? 1 : 0;
  }
}

void multiply_cpu(int n, int* a, int* x, int* y)
{
  for (int i = 0; i < n; i++) {
    x[i] = (a[i]) ? x[i]*y[i] : 0;
  }
}

struct LineItem {
  std::vector<int> l_shipdate;
  std::vector<double> l_quantity;
  std::vector<double> l_extendedprice;
  std::vector<double> l_discount;
  int size;
};

int dtoi(std::string str) {
  std::istringstream date(str);
  std::string time;
  int result = 0;
  int count = 0;
  while (getline(date, time, '-')) {
    int multiplier = 1;
    if (count==1) multiplier=365;
    if (count==2) multiplier=30;
    ++count;
    result += stoi(time)*multiplier;
  }
  return result;
}

void parse_lineitem(std::string path, LineItem& record)
{ 
  std::cout << "Parsing lineitem" << std::endl;
  std::fstream buffer(path);
  std::string line;
  while (getline(buffer, line)) {
    std::istringstream row(line);
    std::string field;
    int column;
    while (getline(row, field, DELIMITER)) {
      if (column==QUANTITY) {
        record.l_quantity.push_back(std::stod(field));
      } else if (column==EXTENDED_PRICE) {
        record.l_extendedprice.push_back(std::stod(field));
      } else if (column==DISCOUNT) {
        record.l_discount.push_back(std::stod(field));
      } else if (column==SHIPDATE) {
        record.l_shipdate.push_back(dtoi(field));
      }
      ++column;
      if (column==NUM_COLUMN) {
        column = 0;
        continue;
      }
    }
    ++record.size;
  }
}

int main(int argc, char** argv)
{
  float r = 1.0;
  parse_lineitem(LINEITEM_PATH);
  if (argc > 1) {
    r = atof(argv[1]);
    std::cout << "Ratio: " << r << std::endl;
  } else { std::cout << "Ratio set to default: " << r << std::endl; }

  std::cout << "Starting program" << std::endl;
  int N = 1<<20;
  int* l_shipdate;
  int* l_quantity;
  int* l_extendedprice;
  int* l_discount;

  // Allocate Unified Memory – accessible from CPU or GPU
  std::cout << "Allocating Memory" << std::endl;
  hipMallocManaged(&l_extendedprice, N*sizeof(int));
  hipMallocManaged(&l_discount, N*sizeof(int));
  hipMallocManaged(&l_quantity, N*sizeof(int));
  hipMallocManaged(&l_shipdate, N*sizeof(int));

  // initialize rng
  std::random_device dev;
  std::mt19937 rng(dev());
  std::uniform_int_distribution<std::mt19937::result_type> generateRandomInt(1, 100);

  // initialize x and y arrays on the host
  std::cout << "Initializing values" << std::endl;
  for (int i = 0; i < N; i++) {
    l_extendedprice[i] = generateRandomInt(rng);
    l_discount[i] = generateRandomInt(rng);
    l_shipdate[i] = generateRandomInt(rng);
    l_quantity[i] = generateRandomInt(rng);
  }

  int N_cpu = N*r;
  int N_gpu = N*(1-r);
  int blockSize = 256;
  int numBlocks = (N_gpu + blockSize - 1) / blockSize;
  std::cout << "cpu:gpu ratio: " << r << ":" << (1-r) << std::endl;

  auto start = std::chrono::steady_clock::now(); 

  std::cout << "Running kernels" << std::endl;
  check_cpu(N_cpu, l_quantity, l_shipdate, l_extendedprice, l_discount);
  check<<<numBlocks, blockSize>>>(N_gpu, l_quantity+N_cpu, l_shipdate+N_cpu, l_extendedprice+N_cpu, l_discount+N_cpu);
  hipDeviceSynchronize();

  multiply_cpu(N_cpu, l_quantity, l_extendedprice, l_discount);
  multiply<<<numBlocks, blockSize>>>(N_gpu, l_quantity+N_cpu, l_extendedprice+N_cpu, l_discount+N_cpu);
  hipDeviceSynchronize();

  auto total = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start).count();
  std::cout << "Total time: " << total << " ms" << std::endl;

  // Read out 'query result'
  int amount = 0;
  for (int i = 0; i < N; i++) if (l_extendedprice[i]) amount++;
  std::cout << "Amount: " << amount << std::endl;
  std::cout << "N: " << N << std::endl;

  // Free memory
  hipFree(l_discount);
  hipFree(l_extendedprice);
  hipFree(l_shipdate);
  hipFree(l_discount);
  
  return 0;
}
