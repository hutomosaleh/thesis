#include <iostream>
#include <fstream>
#include <chrono>
#include <math.h>
#include <random>
#include <sstream>
#include <string>
#include "gpu_kernels.h"

#define QUANTITY 5
#define EXTENDED_PRICE 6
#define DISCOUNT 7
#define SHIPDATE 11
#define NUM_COLUMN 16
#define LINEITEM_PATH "cpp_tpch_q6/data/lineitem.tbl"
#define DELIMITER '|'

void check_cpu(int n, double* l_quantity, int* l_shipdate, double* l_discount)
{
  for (int i = 0; i < n; i++) {
    bool valid_date = (l_shipdate[i] >= 727841 && l_shipdate[i] <= 728206);
    bool valid_quantity = (l_quantity[i] < 24.0);
    bool valid_discount = (l_discount[i] > 0.05 && l_discount[i] < 0.07);
    l_quantity[i] = (valid_date && valid_quantity && valid_discount) ? 1 : 0;
  }
}

void multiply_cpu(int n, double* l_quantity, double* l_extendedprice, double* l_discount)
{
  for (int i = 0; i < n; i++) {
    l_extendedprice[i] = (l_quantity[i]) ? l_extendedprice[i]*l_discount[i] : 0;
  }
}

struct LineItem {
  std::vector<double> l_quantity;
  std::vector<double> l_extendedprice;
  std::vector<double> l_discount;
  std::vector<int> l_shipdate;
  int size = 0;
};

int dtoi(std::string str) {
  std::istringstream date(str);
  std::string time;
  int result = 0;
  int count = 0;
  while (getline(date, time, '-')) {
    int multiplier = 1;
    if (count==1) multiplier=365;
    if (count==2) multiplier=30;
    ++count;
    result += stoi(time)*multiplier;
  }
  return result;
}

void parse_lineitem(std::string path, LineItem& record)
{ 
  std::cout << "Parsing lineitem" << std::endl;
  std::fstream buffer(path);
  std::string line;
  while (getline(buffer, line)) {
    std::istringstream row(line);
    std::string field;
    int column;
    while (getline(row, field, DELIMITER)) {
      if (column==QUANTITY) {
        record.l_quantity.push_back(std::stod(field));
      } else if (column==EXTENDED_PRICE) {
        record.l_extendedprice.push_back(std::stod(field));
      } else if (column==DISCOUNT) {
        record.l_discount.push_back(std::stod(field));
      } else if (column==SHIPDATE) {
        record.l_shipdate.push_back(dtoi(field));
      }
      ++column;
      if (column==NUM_COLUMN) {
        column = 0;
        continue;
      }
    }
    ++record.size;
  }
}

int main(int argc, char** argv)
{
  float r = 1.0;
  LineItem lineitem;
  parse_lineitem(LINEITEM_PATH, lineitem);
  if (argc > 1) {
    r = atof(argv[1]);
    std::cout << "Ratio: " << r << std::endl;
  } else { std::cout << "Ratio set to default: " << r << std::endl; }

  std::cout << "Starting program" << std::endl;
  double* l_quantity = &lineitem.l_quantity[0];
  double* l_extendedprice = &lineitem.l_extendedprice[0];
  double* l_discount = &lineitem.l_discount[0];
  int* l_shipdate = &lineitem.l_shipdate[0];
  int N = lineitem.size;

  // Allocate Unified Memory – accessible from CPU or GPU
  std::cout << "Allocating Memory" << std::endl;
  hipMallocManaged(&l_quantity, N*sizeof(double));
  hipMallocManaged(&l_extendedprice, N*sizeof(double));
  hipMallocManaged(&l_discount, N*sizeof(double));
  hipMallocManaged(&l_shipdate, N*sizeof(int));

  int N_cpu = N*r;
  int N_gpu = N*(1-r);
  int blockSize = 256;
  int numBlocks = (N_gpu + blockSize - 1) / blockSize;
  std::cout << "cpu:gpu ratio: " << r << ":" << (1-r) << std::endl;

  auto start = std::chrono::steady_clock::now(); 

  std::cout << "Running kernels" << std::endl;
  check_cpu(N_cpu, l_quantity, l_shipdate, l_discount);
  check<<<numBlocks, blockSize>>>(N_gpu, l_quantity+N_cpu, l_shipdate+N_cpu, l_discount+N_cpu);
  hipDeviceSynchronize();

  multiply_cpu(N_cpu, l_quantity, l_extendedprice, l_discount);
  multiply<<<numBlocks, blockSize>>>(N_gpu, l_quantity+N_cpu, l_extendedprice+N_cpu, l_discount+N_cpu);
  hipDeviceSynchronize();

  auto total = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start).count();
  std::cout << "Total time: " << total << " ms" << std::endl;

  // Read out 'query result'
  int amount = 0;
  for (int i = 0; i < N; i++) if (l_extendedprice[i]) amount++;
  std::cout << "Amount: " << amount << std::endl;
  std::cout << "N: " << N << std::endl;

  // Free memory
  hipFree(l_discount);
  hipFree(l_extendedprice);
  hipFree(l_shipdate);
  hipFree(l_discount);
  
  return 0;
}
