
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <random>

// Kernel function to multiply the elements of two arrays
__global__
void multiply(int n, int* a, int* x, int* y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    // If WHERE is true, multiply l_extendedprice & l_discount
    x[i] = (a[i]) ? x[i]*y[i] : 0;
  }
}

// Kernel function to check condition
__global__
void check(int n, int* a, int* b, int* c, int* d) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i+= stride) {
    // Check if WHERE is true, put result in l_quantity
    bool condition = (a[i]>50 && b[i]>50 && c[i]>50 && d[i]>50); // Mock condition
    a[i] = condition ? 1 : 0;
  }
}

// CPU function
void check_cpu(int n, int* a, int* b, int* c, int* d) {
  for (int i = 0; i < n; i++) {
    bool condition = (a[i]>50 && b[i]>50 && c[i]>50 && d[i]>50); // Mock condition
    a[i] = condition ? 1 : 0;
  }
}

int main(void)
{
  std::cout << "Starting program" << std::endl;
  int N = 1<<20;
  int* l_shipdate;
  int* l_quantity;
  int* l_extendedprice;
  int* l_discount;

  // Allocate Unified Memory – accessible from CPU or GPU
  std::cout << "Allocating Memory" << std::endl;
  hipMallocManaged(&l_extendedprice, N*sizeof(int));
  hipMallocManaged(&l_discount, N*sizeof(int));
  hipMallocManaged(&l_quantity, N*sizeof(int));
  hipMallocManaged(&l_shipdate, N*sizeof(int));

  // initialize rng
  std::random_device dev;
  std::mt19937 rng(dev());
  std::uniform_int_distribution<std::mt19937::result_type> generateRandomInt(1, 100);

  // initialize x and y arrays on the host
  std::cout << "Initializing values" << std::endl;
  for (int i = 0; i < N; i++) {
    l_extendedprice[i] = generateRandomInt(rng);
    l_discount[i] = generateRandomInt(rng);
    l_shipdate[i] = generateRandomInt(rng);
    l_quantity[i] = generateRandomInt(rng);
  }

  // Run kernel on 1M elements on the GPU
  std::cout << "Running kernels" << std::endl;
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  check_cpu(N, l_quantity, l_shipdate, l_extendedprice, l_discount);

  //check<<<numBlocks, blockSize>>>(N, l_quantity, l_shipdate, l_extendedprice, l_discount);
  // cudaDeviceSynchronize();  // Is this necessary?

  multiply<<<numBlocks, blockSize>>>(N, l_quantity, l_extendedprice, l_discount);
  hipDeviceSynchronize();

  // Read out 'query result'
  int amount = 0;
  for (int i = 0; i < N; i++) if (l_extendedprice[i]) amount++;
  std::cout << "Amount: " << amount << std::endl;
  std::cout << "N: " << N << std::endl;

  // Free memory
  hipFree(l_discount);
  hipFree(l_extendedprice);
  hipFree(l_shipdate);
  hipFree(l_discount);
   
  return 0;
}
