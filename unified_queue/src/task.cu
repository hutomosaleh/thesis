#include "task.h"

#include <atomic>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <vector>

#include "defs.hpp"
#include "data_types.hpp"
#include "cpu_kernels.hpp"
#include "gpu_kernels.h"

void Task::set_id(int id)
{
  _id = id;
}

void Task::add(TupleQ6 tuple)
{
  _data.push_back(tuple);
  ++_size;
}

void Task::consume(int type, hipStream_t* streams)
{
  double* q;
  double* e;
  double* d;
  int* s;

  switch (type)
  {
#ifdef MALLOCHOST
    case CPU_TASK:

      hipHostMalloc(&q, _size*sizeof(double));
      hipHostMalloc(&e, _size*sizeof(double));
      hipHostMalloc(&d, _size*sizeof(double));
      hipHostMalloc(&s, _size*sizeof(int));
      for(int i=0; i<_size; ++i)
      {
        q[i] = _data[i].quantity;
        e[i] = _data[i].extendedprice;
        d[i] = _data[i].discount;
        s[i] = _data[i].shipdate;
      }
      check_cpu(_size, q, s, d);
      multiply_cpu(_size, q, e, d);
      for (int i = 0; i < _size; i++)
      {
        if (e[i])
        {
          _hits++;
          _result += e[i];
        }
      }
      hipHostFree(q);
      hipHostFree(e);
      hipHostFree(s);
      hipHostFree(d);

      break;
#else
    case CPU_TASK:

      q = (double*) malloc(_size*sizeof(double));
      e = (double*) malloc(_size*sizeof(double));
      d = (double*) malloc(_size*sizeof(double));
      s = (int*) malloc(_size*sizeof(int));
      for(int i=0; i<_size; ++i)
      {
        q[i] = _data[i].quantity;
        e[i] = _data[i].extendedprice;
        d[i] = _data[i].discount;
        s[i] = _data[i].shipdate;
      }
      check_cpu(_size, q, s, d);
      multiply_cpu(_size, q, e, d);
      for (int i = 0; i < _size; i++)
      {
        if (e[i])
        {
          _hits++;
          _result += e[i];
        }
      }
      free(q);
      free(e);
      free(s);
      free(d);

      break;
#endif
#ifdef MALLOCMANAGED
    case GPU_TASK:

      // Allocate device memory
      hipMallocManaged(&q, _size*sizeof(double));
      hipMallocManaged(&e, _size*sizeof(double));
      hipMallocManaged(&d, _size*sizeof(double));
      hipMallocManaged(&s, _size*sizeof(int));
      for(int i=0; i<_size; ++i)
      {
        q[i] = _data[i].quantity;
        e[i] = _data[i].extendedprice;
        d[i] = _data[i].discount;
        s[i] = _data[i].shipdate;
      }

      int block_number = (_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
      check<<<block_number, BLOCK_SIZE>>>(_size, q, s, d);
      multiply<<<block_number, BLOCK_SIZE>>>(_size, q, e, d);

      // Compare for query hits and update results
      for (int i = 0; i < _size; i++)
      {
        if (e[i])
        {
          _hits++;
          _result += e[i];
        }
      }

      // Release memory
      hipFree(q);
      hipFree(e);
      hipFree(s);
      hipFree(d);
      break;
#else
    case GPU_TASK:

      // Allocate host memory
      double* q_h = (double*)malloc(_size*sizeof(double));
      double* e_h = (double*)malloc(_size*sizeof(double));
      double* d_h = (double*)malloc(_size*sizeof(double));
      int* s_h = (int*)malloc(_size*sizeof(int));
      
      // Allocate device memory
      hipMalloc(&q, _size*sizeof(double));
      hipMalloc(&e, _size*sizeof(double));
      hipMalloc(&d, _size*sizeof(double));
      hipMalloc(&s, _size*sizeof(int));
      for(int i=0; i<_size; ++i)
      {
        q_h[i] = _data[i].quantity;
        e_h[i] = _data[i].extendedprice;
        d_h[i] = _data[i].discount;
        s_h[i] = _data[i].shipdate;
      }

      if (streams != nullptr)
      {
        const int stream_size = _size / STREAM_NUM;
        for (int i=0; i<STREAM_NUM; i++)
        {
          int offset = i * stream_size;
          // Copy host to device
          hipMemcpyAsync(&q[offset], &q_h[offset], stream_size*sizeof(double), hipMemcpyHostToDevice, streams[i]);
          hipMemcpyAsync(&e[offset], &e_h[offset], stream_size*sizeof(double), hipMemcpyHostToDevice, streams[i]);
          hipMemcpyAsync(&d[offset], &d_h[offset], stream_size*sizeof(double), hipMemcpyHostToDevice, streams[i]);
          hipMemcpyAsync(&s[offset], &s_h[offset], stream_size*sizeof(int), hipMemcpyHostToDevice, streams[i]);
          check<<<stream_size/BLOCK_SIZE, BLOCK_SIZE, 0, streams[i]>>>(_size, q, s, d);
          multiply<<<stream_size/BLOCK_SIZE, BLOCK_SIZE, 0, streams[i]>>>(_size, q, e, d);
        }
        for (int i = 0; i < STREAM_NUM; ++i)
        {
          int offset = i * stream_size;
          hipMemcpyAsync(&e_h[offset], &e[offset], stream_size*sizeof(double), hipMemcpyDeviceToHost, streams[i]);
        }
      }
      else {
        int block_number = (_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        // Copy host to device
        hipMemcpy(q, q_h, _size*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(e, e_h, _size*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d, d_h, _size*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(s, s_h, _size*sizeof(int), hipMemcpyHostToDevice);

        // Run kernels
        check<<<block_number, BLOCK_SIZE>>>(_size, q, s, d);
        multiply<<<block_number, BLOCK_SIZE>>>(_size, q, e, d);

        // Copy device result to host
        hipMemcpy(e_h, e, _size*sizeof(double), hipMemcpyDeviceToHost);
      }

      // Compare for query hits and update results
      for (int i = 0; i < _size; i++)
      {
        if (e_h[i])
        {
          _hits++;
          _result += e_h[i];
        }
      }

      // Release memory
      hipFree(q);
      hipFree(e);
      hipFree(s);
      hipFree(d);
      free(q_h);
      free(e_h);
      free(s_h);
      free(d_h);

      break;
#endif
  }
}

int Task::get_hits()
{
  return _hits;
}

double Task::get_result()
{
  return _result;
}
