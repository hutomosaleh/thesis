#include "task.h"

#include <atomic>
#include <iostream>
#include <vector>

#include "defs.hpp"
#include "data_types.hpp"
#include "cpu_kernels.hpp"
#include "gpu_kernels.h"

void Task::set_id(int id)
{
  _id = id;
}

void Task::add(TupleQ6 tuple)
{
  _data.push_back(tuple);
  ++_size;
}

void Task::consume(int type)
{
  double* q;
  double* e;
  double* d;
  int* s;

  switch (type)
  {
    case CPU_TASK:

      hipHostMalloc(&q, _size*sizeof(double));
      hipHostMalloc(&e, _size*sizeof(double));
      hipHostMalloc(&d, _size*sizeof(double));
      hipHostMalloc(&s, _size*sizeof(int));
      for(int i=0; i<_size; ++i)
      {
        q[i] = _data[i].quantity;
        e[i] = _data[i].extendedprice;
        d[i] = _data[i].discount;
        s[i] = _data[i].shipdate;
      }
      check_cpu(_size, q, s, d);
      multiply_cpu(_size, q, e, d);
      for (int i = 0; i < _size; i++)
      {
        if (e[i])
        {
          _hits++;
          _result += e[i];
        }
      }
      hipFree(q);
      hipFree(e);
      hipFree(s);
      hipFree(d);

      break;

    case GPU_TASK:

      // Allocate host memory
      double* q_h = (double*)malloc(_size*sizeof(double));
      double* e_h = (double*)malloc(_size*sizeof(double));
      double* d_h = (double*)malloc(_size*sizeof(double));
      int* s_h = (int*)malloc(_size*sizeof(int));
      
      // Allocate device memory
      hipMalloc(&q, _size*sizeof(double));
      hipMalloc(&e, _size*sizeof(double));
      hipMalloc(&d, _size*sizeof(double));
      hipMalloc(&s, _size*sizeof(int));
      for(int i=0; i<_size; ++i)
      {
        q_h[i] = _data[i].quantity;
        e_h[i] = _data[i].extendedprice;
        d_h[i] = _data[i].discount;
        s_h[i] = _data[i].shipdate;
      }

      // Copy host to device
      hipMemcpy(q, q_h, _size*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(e, e_h, _size*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d, d_h, _size*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(s, s_h, _size*sizeof(int), hipMemcpyHostToDevice);

      int blockSize = 128;
      int numBlocks = (_size + blockSize - 1) / blockSize;
      check<<<numBlocks, blockSize>>>(_size, q, s, d);
      multiply<<<numBlocks, blockSize>>>(_size, q, e, d);
      hipDeviceSynchronize();

      // Copy device result to host
      hipMemcpy(e_h, e, _size*sizeof(double), hipMemcpyDeviceToHost);

      // Compare for query hits and update results
      for (int i = 0; i < _size; i++)
      {
        if (e_h[i])
        {
          _hits++;
          _result += e_h[i];
        }
      }

      // Release memory
      hipFree(q);
      hipFree(e);
      hipFree(s);
      hipFree(d);
      free(q_h);
      free(e_h);
      free(s_h);
      free(d_h);

      break;
  }
}

int Task::get_hits()
{
  return _hits;
}

double Task::get_result()
{
  return _result;
}
